#include "hip/hip_runtime.h"
#include "GPUCompute.cuh"

__global__ void vectorAdd(int* a, int* b, int* c) 
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	return;
}



std::tuple<std::vector<std::tuple<unsigned char, unsigned char, unsigned char>>, std::vector<double>> GPUCompute::Compute(std::vector<unsigned char> pBuf, int ref_size, int game_running) {
	//vector for our reference with only RGB not Alpha channel included
	std::vector<std::tuple<unsigned char, unsigned char, unsigned char>> ref_vector{};

	std::vector<double> position_vector{};

	/*OutputDebugStringA("\nHello from GPUCOMPUTE\n");
	std::string str = "\nHello from GPUCOMPUTE\n";
	OutputDebugStringA(str.c_str());*/

	int a[] = { 1,2,3 };
	int b[] = { 4,5,6 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	//pointers to the gpu
	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	//allocate memory in the GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	//copy the vectors into the GPU
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
		
	//vectorAdd<<<GRID_SIZE, BLOCK_SIZE = number of vectors>>>
	vectorAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	try {
		for (int i = 0; i < (ref_size - 4); i++) {
			unsigned char r = pBuf.at((i * 4) + 2);
			unsigned char g = pBuf.at((i * 4) + 1);
			unsigned char b = pBuf.at(i * 4);
			/*
			if (g > 85 && r < g/2 && b < g/2) {
				std::string temp = "{" + to_string(r) + "," +
					to_string(g) + "," + to_string(b) + "}";
				if (!(colors.find(temp) != std::string::npos)) {
					colors += temp + ",";
				};

			}*/


			//vector<GPUCompute::_RGB> found_colors{};
			std::tuple<unsigned char, unsigned char, unsigned char> rgb;
			if (game_running == 1) {
				for (std::tuple<unsigned char, unsigned char, unsigned char> rgbt : GPUCompute::cs_colors) {
					if (r == std::get<0>(rgbt) && g == std::get<1>(rgbt) && b == std::get<2>(rgbt)) {
						rgb = std::make_tuple(r, g, b);
						int c = 0;
						for (std::tuple<unsigned char, unsigned char, unsigned char> vec_rgb : ref_vector) {
							if (r == std::get<0>(vec_rgb) && g == std::get<1>(vec_rgb) && b == std::get<2>(vec_rgb)) {
								c++;
								continue;
							}
						}
						if (c == 0) {
							ref_vector.push_back(rgb);
							position_vector.push_back(i);
						}
					}
				}
			}
			else {
				for (std::tuple<unsigned char, unsigned char, unsigned char> rgbt : GPUCompute::dota_colors) {
					if (r == std::get<0>(rgbt) && g == std::get<1>(rgbt) && b == std::get<2>(rgbt)) {
						rgb = std::make_tuple(r, g, b);
						int c = 0;
						for (std::tuple<unsigned char, unsigned char, unsigned char> vec_rgb : ref_vector) {
							if (r == std::get<0>(vec_rgb) && g == std::get<1>(vec_rgb) && b == std::get<2>(vec_rgb)) {
								c++;
								continue;
							}
						}
						if (c == 0) {
							ref_vector.push_back(rgb);
							position_vector.push_back(i);
						}
					}
				}
			}
		}
	}
	catch (std::exception e) {
		std::string test = e.what();
		_log::log("\nERROR -SC");
		_log::log(test);
		_log::log("\nERROR -SC");
	}
	return std::make_tuple(ref_vector, position_vector);
};