#include "hip/hip_runtime.h"
#include "GPUCompute.cuh"



// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(unsigned char* template_vector, unsigned char* sshot_vector, int* match_count, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;


    // Make sure we do not go out of bounds
    if (id < n) 
    {
        if (template_vector[id] - sshot_vector[id] > 250 ||
            template_vector[id] - sshot_vector[id] < 6)
        {
            //TODO CONCURRENCY IS FAILING ME HARD HERE> need to figure
            //++match_count[id];
        }
    }
}



int GPUCompute::Compute(std::vector<  unsigned char> pBuf, int ref_size, int game_running) {
	//vector for our reference with only RGB not Alpha channel included
	std::vector<std::tuple<  char,   char,   char>> ref_vector{ {1,2,3} };

	std::vector<double> position_vector{5,5,5,5,5};

    // Size of vectors
    int pixels_times_rgb = sizeof(dota_colors) ;

    // Host input vectors
    unsigned char* templ_array_host = nullptr;
    unsigned char* sshot_array_host = nullptr;
    //Host output vector
    int* match_count_host = nullptr;

    // Device input vectors
    unsigned char* templ_array_gpu = nullptr;
    unsigned char* sshot_array_gpu = nullptr;
    //Device output vector
    int* match_count_gpu = nullptr;

    // Allocate memory for each vector on host
    templ_array_host = (unsigned char*)malloc(pixels_times_rgb * sizeof(unsigned char*));
    sshot_array_host = (unsigned char*)malloc(pixels_times_rgb * sizeof(unsigned char*));
    match_count_host = (int*)malloc(pixels_times_rgb * sizeof(int*));

    // Allocate memory for each vector on GPU
    hipMalloc(&templ_array_gpu, pixels_times_rgb * sizeof(unsigned char*));
    hipMalloc(&sshot_array_gpu, pixels_times_rgb * sizeof(unsigned char*));
    hipMalloc(&match_count_gpu, pixels_times_rgb * sizeof(int*));
    //image dimensions in pixels
    const int width_screenshot = 1920;
    const int width_template = 321;
    const int height_screenshot = 1080;
    const int height_template = 65;

    // outside loop is iterating the height
    // adding the width of the screenshot means that we are going to the next height pixel
    for (int height_iterator = 1; height_iterator < height_screenshot +1; height_iterator++)
    {
        for (int width_iterator = 1; width_iterator < width_screenshot +1; width_iterator++)
        {
            int sizeofpbuf = pBuf.capacity();
            int screenshot_iterator = 0;
            int alpha_iterator = 0;

            for (int template_iterator = 0; template_iterator < (sizeof(dota_colors) / sizeof(int)); template_iterator++)
            {
                //int testt = (sizeof(dota_colors) / sizeof(int) - 3000);
                alpha_iterator++;
             
                templ_array_host[template_iterator] = dota_colors[template_iterator];

                //removes the alpha layer from the equation aka every fourth increment
                if (alpha_iterator > 3) {
                    alpha_iterator = 0;
                    screenshot_iterator++;
                }

                sshot_array_host[template_iterator] = pBuf.at(screenshot_iterator);
                //for checking the values during debug
                unsigned char xxxxx = sshot_array_host[screenshot_iterator];
                //this means we are at the end of the line of the template image
                //move the sshot to the next line
                double testing = template_iterator % width_template;
                if (template_iterator % width_template == 0/*this just means we are at the end of the line of pixels the size of template width
                    in the screenshot wich is a small percentage of the size of the screenshot*/ 
                    && template_iterator != 0/*first line will start at zero so ignore it*/)
                {
                    //width screenshot + current position minus width of template equals the next line in the 'grid' of pixels
                    screenshot_iterator = ((width_screenshot * 4) + screenshot_iterator) - width_template;
                }
                else
                {
                    screenshot_iterator++;
                }
            }
            // Copy host vectors to device
            hipMemcpy(templ_array_gpu, templ_array_host, pixels_times_rgb * sizeof(unsigned char*), hipMemcpyHostToDevice);
            hipMemcpy(sshot_array_gpu, sshot_array_host, pixels_times_rgb * sizeof(unsigned char*), hipMemcpyHostToDevice);
            

            int blockSize, gridSize;

            // Number of threads in each thread block
            blockSize = 1024;

            // Number of thread blocks in grid
            gridSize = (int)ceil((float)pixels_times_rgb / blockSize);

            // Execute the kernel
            vecAdd <<< gridSize, blockSize >>> (templ_array_gpu, sshot_array_gpu, match_count_gpu, pixels_times_rgb);

            // Copy array back to host
            hipMemcpy(match_count_host, match_count_gpu, sizeof(int*) * pixels_times_rgb, hipMemcpyDeviceToHost);

            int TEST0 = match_count_host[0];
            int TEST1 = sizeof(match_count_host[0]);
            int TEST2 = match_count_host[500];
            
            ++match_count_host[0];
            int TEST3 = sizeof(match_count_host[0]);
            if (TEST0 > 1)
            {
                //hello
                printf("test");
            }

            // Release device memory
            hipFree(templ_array_gpu);
            hipFree(sshot_array_gpu);
            hipFree(match_count_gpu);

        }

    }



    // Release host memory
    free(templ_array_host);
    free(sshot_array_host);
    free(match_count_host);
	

	//try {
	//	for (int i = 0; i < (ref_size - 4); i++) {
	//		  char r = pBuf.at((i * 4) + 2);
	//		  char g = pBuf.at((i * 4) + 1);
	//		  char b = pBuf.at(i * 4);
	//		/*
	//		if (g > 85 && r < g/2 && b < g/2) {
	//			std::string temp = "{" + to_string(r) + "," +
	//				to_string(g) + "," + to_string(b) + "}";
	//			if (!(colors.find(temp) != std::string::npos)) {
	//				colors += temp + ",";
	//			};

	//		}*/
	//		

	//		//vector<GPUCompute::_RGB> found_colors{};
	//		std::tuple<  char,   char,   char> rgb;
	//		
	//		for (std::tuple<  char,   char,   char> rgbt : cs_colors) {
	//			if (r == std::get<0>(rgbt) && g == std::get<1>(rgbt) && b == std::get<2>(rgbt)) {
	//				rgb = std::make_tuple(r, g, b);
	//				int c = 0;
	//				for (std::tuple<  char,   char,   char> vec_rgb : ref_vector) {
	//					if (r == std::get<0>(vec_rgb) && g == std::get<1>(vec_rgb) && b == std::get<2>(vec_rgb)) {
	//						c++;
	//						continue;
	//					}
	//				}
	//				if (c == 0) {
	//					ref_vector.push_back(rgb);
	//					position_vector.push_back(i);
	//				}
	//			}
	//		}
	//		
	//	}
	//}
	//catch (std::exception e) {
	//	std::string test = e.what();
	//	_log::log("\nERROR -SC");
	//	_log::log(test);
	//	_log::log("\nERROR -SC");
	//}
    return 0;
};