#include "hip/hip_runtime.h"
#include <stdio.h>
#include <debugapi.h>
#include "GPUCompute.cuh"
#include "_log.h"

namespace GPUCompute
{
	std::tuple<std::vector<_RGB>, std::vector<double>> Compute(std::vector<BYTE> pBuf, int ref_size, int game_running) {
		OutputDebugStringA("\nHello from GPUCOMPUTE\n");
		std::string str = "\nHello from GPUCOMPUTE\n";
		OutputDebugStringA(str.c_str());



		try {
			for (int i = 0; i < (ref_size - 4); i++) {
				unsigned char r = pBuf.at((i * 4) + 2);
				unsigned char g = pBuf.at((i * 4) + 1);
				unsigned char b = pBuf.at(i * 4);
				/*
				if (g > 85 && r < g/2 && b < g/2) {
					std::string temp = "{" + to_string(r) + "," +
						to_string(g) + "," + to_string(b) + "}";
					if (!(colors.find(temp) != std::string::npos)) {
						colors += temp + ",";
					};

				}*/


				//vector<_RGB> found_colors{};
				_RGB rgb{ NULL };
				if (game_running == 1) {
					for (_RGB rgbt : cs_colors) {
						if (r == rgbt.r && g == rgbt.g && b == rgbt.b) {
							rgb.r = r;
							rgb.g = g;
							rgb.b = b;
							int c = 0;
							for (_RGB vec_rgb : ref_vector) {
								if (r == vec_rgb.r && g == vec_rgb.g && b == vec_rgb.b) {
									c++;
									continue;
								}
							}
							if (c == 0) {
								ref_vector.push_back(rgb);
								position_vector.push_back(i);
							}
						}
					}
				}
				else {
					for (_RGB rgbt : dota_colors) {
						if (r == rgbt.r && g == rgbt.g && b == rgbt.b) {
							rgb.r = r;
							rgb.g = g;
							rgb.b = b;
							int c = 0;
							for (_RGB vec_rgb : ref_vector) {
								if (r == vec_rgb.r && g == vec_rgb.g && b == vec_rgb.b) {
									c++;
									continue;
								}
							}
							if (c == 0) {
								ref_vector.push_back(rgb);
								position_vector.push_back(i);
							}
						}
					}
				}
			}
		}
		catch (std::exception e) {
			std::string test = e.what();
			_log::log("\nERROR -SC");
			_log::log(test);
			_log::log("\nERROR -SC");
		}
		return std::make_tuple(ref_vector, position_vector);
	};

}